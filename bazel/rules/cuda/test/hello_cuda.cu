///
/// @file
/// @copyright Copyright (c) 2022. NVIDIA Corporation. All Rights Reserved.
///

#include <hip/hip_runtime.h>
#include <iostream>

__global__ void Kernel()
{
    printf("Hello CUDA!\n");
}

int main()
{
    Kernel<<<1, 1>>>();
    hipDeviceSynchronize();
    return 0;
}
